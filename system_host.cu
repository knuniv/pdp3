#include "hip/hip_runtime.h"
//#include <cutil.h>				// cutil32.lib
#include <string.h>
#include "system_kern.cu"


extern Particles_struct specie;
	
extern "C"
{

int iDivUp (int a, int b)
{
   return (a % b != 0) ? (a / b + 1) : (a / b);
}
void computeNumBlocks (int numPnts, int maxThreads, int &numBlocks, int &numThreads)
{

    //numThreads = min( maxThreads, numPnts );
	numThreads = 256;
    numBlocks = iDivUp ( numPnts, numThreads );
}

bool InitCUDA(void)
{
   int count = 0;
   int i = 0;

   hipGetDeviceCount(&count);
   if(count == 0) {
           fprintf(stderr, "There is no device.\n");
           return false;
   }

   for(i = 0; i < count; i++) {
           hipDeviceProp_t prop;
           if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
                   if(prop.major >= 1) {
                           break;
                   }
           }
   }
   if(i == count) {
           fprintf(stderr, "There is no device supporting CUDA.\n");
           return false;
   }
   hipSetDevice(i);

   printf("CUDA initialized.\n");
   return true;
}


void SetupCUDA (int grid_num1, int grid_num3, int number)
{
  //computeNumBlocks ( params.NumParticles, 256, params.NumBlocks, params.NumThreads);			// particles
  //computeNumBlocks ( params.NumBoundaries, 256, params.NumBoundaryBlocks, params.NumBoundaryThreads);			// boundaries
 // CUDA_SAFE_CALL ( hipMalloc ( (void**) &&cuda_specie, sizeof(Particles_struct)) );	
  //hipMemcpyToSymbol ( (char *)&cuda_specie, &specie, sizeof(Particles_struct), 0, hipMemcpyHostToDevice ) ;
  hipMalloc ( (void**) &x1, sizeof(flcuda)* number) ;
  hipMalloc ( (void**) &x3, sizeof(flcuda)* number) ;
  hipMalloc ( (void**) &v1, sizeof(flcuda)* number) ;
  hipMalloc ( (void**) &v2, sizeof(flcuda)* number) ;
  hipMalloc ( (void**) &v3, sizeof(flcuda)* number) ;	
  hipMalloc ( (void**) &is_alive, sizeof(bool)* number) ;	

  hipMalloc ( (void**) &e1, sizeof(flcuda)* (grid_num1 - 1) * grid_num3) ;
  hipMalloc ( (void**) &e2, sizeof(flcuda)* grid_num1 * grid_num3 ) ;
  hipMalloc ( (void**) &e3, sizeof(flcuda)* grid_num1 * (grid_num3 - 1) ) ;	

  hipMalloc ( (void**) &h1, sizeof(flcuda)* grid_num1 * (grid_num3 - 1) ) ;
  hipMalloc ( (void**) &h2, sizeof(flcuda)* (grid_num1 - 1) * (grid_num3 - 1) ) ;
  hipMalloc ( (void**) &h3, sizeof(flcuda)* (grid_num1 - 1) * grid_num3) ;	



  hipDeviceSynchronize ();
}

void CopySpecie2Cuda (Particles_struct specie)
{
  hipMemcpyToSymbol ( (char *)&cuda_specie, &specie, sizeof(Particles_struct), 0, hipMemcpyHostToDevice ) ;
  hipDeviceSynchronize ();
}

//void TransferXVToCUDA (flcuda* CPU_x1, flcuda* CPU_x3, flcuda* CPU_v1, flcuda* CPU_v2, flcuda* CPU_v3);
//void TransferEHToCUDA (flcuda* CPU_e1, flcuda* CPU_e2, flcuda* CPU_e3, flcuda* CPU_h1, flcuda* CPU_h2, flcuda* CPU_h3);
//void TransferXVFromCUDA (flcuda* CPU_x1, flcuda* CPU_x3, flcuda* CPU_v1, flcuda* CPU_v2, flcuda* CPU_v3);

void TransferXVToCUDA (flcuda* CPU_x1, flcuda* CPU_x3, flcuda* CPU_v1, flcuda* CPU_v2, 
					   flcuda* CPU_v3, bool* CPU_is_alive, int number)
{
     hipMemcpy (x1, CPU_x1, number * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (x3, CPU_x3, number * sizeof(flcuda), hipMemcpyHostToDevice ) ;

	 hipMemcpy (v1, CPU_v1, number * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (v2, CPU_v2, number * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (v3, CPU_v3, number * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (is_alive, CPU_is_alive, number * sizeof(bool), hipMemcpyHostToDevice ) ;
	
	hipDeviceSynchronize ();
}

void TransferEHToCUDA (flcuda* CPU_e1, flcuda* CPU_e2, flcuda* CPU_e3, flcuda* CPU_h1, flcuda* CPU_h2, flcuda* CPU_h3, int grid_num1, int grid_num3)
{
     hipMemcpy (e1, CPU_e1, (grid_num1 - 1) * grid_num3 * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (e2, CPU_e2, grid_num1 * grid_num3 * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (e3, CPU_e3, grid_num1 * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyHostToDevice ) ;

	 hipMemcpy (h1, CPU_h1, grid_num1 * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (h2, CPU_h2, (grid_num1 - 1) * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	 hipMemcpy (h3, CPU_h3, (grid_num1 - 1) * grid_num3 * sizeof(flcuda), hipMemcpyHostToDevice ) ;
	
	hipDeviceSynchronize ();
}

void TransferEHFromCUDA (flcuda* CPU_e1, flcuda* CPU_e2, flcuda* CPU_e3, flcuda* CPU_h1, flcuda* CPU_h2, flcuda* CPU_h3, int grid_num1, int grid_num3)
{
     hipMemcpy (CPU_e1, e1, (grid_num1 - 1) *  grid_num3 *      sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_e2, e2,  grid_num1      *  grid_num3 *      sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_e3, e3,  grid_num1      * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyDeviceToHost ) ;

	 hipMemcpy (CPU_h1, h1,  grid_num1      * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_h2, h2, (grid_num1 - 1) * (grid_num3 - 1) * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_h3, h3, (grid_num1 - 1) *  grid_num3 *      sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	
	hipDeviceSynchronize ();
}

void TransferXVFromCUDA (flcuda* CPU_x1, flcuda* CPU_x3, flcuda* CPU_v1, flcuda* CPU_v2, flcuda* CPU_v3, int number)
{
  //   hipMemcpy (CPU_x1, x1, number * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 //hipMemcpy (CPU_x3, x3, number * sizeof(flcuda), hipMemcpyDeviceToHost ) ;

	 hipMemcpy (CPU_v1, v1, number * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_v2, v2, number * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	 hipMemcpy (CPU_v3, v3, number * sizeof(flcuda), hipMemcpyDeviceToHost ) ;
	
	hipDeviceSynchronize ();
}

//void TransferToCUDA (Particle* CPU_ParticlesArray, int numPoints )
//{
//    CUDA_SAFE_CALL( hipMemcpy (ParticlesArray, CPU_ParticlesArray, numPoints * sizeof(Particle), hipMemcpyHostToDevice ) );
//	hipDeviceSynchronize ();
//}
//
//void TransferFromCUDA ( Particle* CPU_ParticlesArray, int numPoints )
//{
//	CUDA_SAFE_CALL( hipMemcpy ( CPU_ParticlesArray, ParticlesArray, numPoints * sizeof(Particle), hipMemcpyDeviceToHost ) );
//	hipDeviceSynchronize ();		
//}

void CUDA_StepV(int number, flcuda dt)
{
	int numBlocks = 0, numThreads = 0;
	//computeNumBlocks (cuda_specie.number, 256, numBlocks, numThreads);
	numThreads = 240;
	numBlocks = iDivUp(number, numThreads);
	StepV<<<numBlocks, numThreads>>> (x1, x3, v1, v2, v3, e1, e2, e3, 
		                              h1, h2, h3, is_alive, number, dt);	
	//CUT_CHECK_ERROR( "Kernel execution failed");
	hipDeviceSynchronize ();
}

//void TransferFromCUDA ( Particle* CPU_ParticlesArray, int numPoints )
//{
//	CUDA_SAFE_CALL( hipMemcpy ( CPU_ParticlesArray, ParticlesArray, numPoints * sizeof(Particle), hipMemcpyDeviceToHost ) );
//	hipDeviceSynchronize ();		
//}



//void CUDA_Advance (flcuda dt)
//{
//	Advance<<< params.NumBlocks, params.NumThreads>>> (ParticlesArray, params.TimeStep);	
//	CUT_CHECK_ERROR( "Kernel execution failed");
//	hipDeviceSynchronize ();
//}
//
//void CUDA_AdvanceCoordinates ()
//{
//	AdvanceCoordinates<<< params.NumBlocks, params.NumThreads>>> (ParticlesArray, params.TimeStep);	
//	CUT_CHECK_ERROR( "Kernel execution failed");
//	hipDeviceSynchronize ();
//}
//void CUDA_AdvanceVelocities ()
//{
//	AdvanceVelocities<<< params.NumBlocks, params.NumThreads>>> (ParticlesArray, params.TimeStep);	
//	CUT_CHECK_ERROR( "Kernel execution failed");
//	hipDeviceSynchronize ();
//}
//
//
//void CUDA_CopyParticlesToGLBuffer(float3* positions)
//{
//	CopyParticlesToGLBuffer<<< params.NumBlocks, params.NumThreads>>> (ParticlesArray, positions );
//}

}