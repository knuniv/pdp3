#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "system_kern.cuh"

#define TOTAL_THREADS		65536
#define BLOCK_THREADS		256
	
__device__ Particles_struct cuda_specie;    // simulation data (on device)
__device__ flcuda* x1;
__device__ flcuda* x3;
__device__ flcuda* v1;
__device__ flcuda* v2;
__device__ flcuda* v3;
//electric and magnetic field
__device__ flcuda* e1;
__device__ flcuda* e2;
__device__ flcuda* e3;

__device__ flcuda* h1;
__device__ flcuda* h2;
__device__ flcuda* h3;


//current and charge density
__device__ flcuda* cur1;
__device__ flcuda* cur2;
__device__ flcuda* cur3;
__device__ flcuda* rho;

//is alive array
__device__ bool* is_alive;



__global__ void StepV(flcuda* x1, flcuda* x3, flcuda* v1, flcuda* v2, flcuda* v3, 
					  flcuda* e1, flcuda* e2, flcuda* e3, flcuda* h1, flcuda* h2, 
					  flcuda* h3, bool* is_alive, int number, flcuda timestep)
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	flcuda gamma, gamma_inv;
	flcuda e1_val = 0.0, e2_val, e3_val, b1_val, b2_val, b3_val, vv1, vv2, vv3;
	const flcuda mu0 = (flcuda)1.0e-6;
	flcuda const1 = cuda_specie.charge*timestep/(flcuda)2.0/cuda_specie.mass;
	flcuda const2;
	//if (t->current_time == t->start_time) const1 = const1/2.0;
		if (is_alive[i] && (i < number))
		{
	   	      e1_val = get_field_e(x1[i], x3[i], e1, 1, cuda_specie) * const1;
	          e2_val = get_field_e(x1[i], x3[i], e2, 2, cuda_specie) * const1;
	          e3_val = get_field_e(x1[i], x3[i], e3, 3, cuda_specie) * const1;



	        b1_val = get_field_h(x1[i] ,x3[i], h1, 1, cuda_specie)*mu0*const1;
	        b2_val = get_field_h(x1[i], x3[i], h2, 2, cuda_specie)*mu0*const1;
	        b3_val = get_field_h(x1[i], x3[i], h3, 3, cuda_specie)*mu0*const1;



			////1. Multiplication by relativistic factor
			////u(n-1/2) = gamma(n-1/2)*v(n-1/)
			gamma = get_gamma(i, v1, v2, v3);
			v1[i] = gamma*v1[i];
			v2[i] = gamma*v2[i];
			v3[i] = gamma*v3[i];

			//2. Half acceleration in the electric field
			//u'(n) = u(n-1/2) + q*dt/2/m*E(n)

			v1[i] = v1[i] + e1_val;
			v2[i] = v2[i] + e2_val;
			v3[i] = v3[i] + e3_val;

			//3. Rotation in the magnetic field
			//u" = u' + 2/(1+B'^2)[(u' + [u'xB'(n)])xB'(n)]
			//B'(n) = B(n)*q*dt/2/mass/gamma(n)
			gamma_inv = get_gamma_inv(i, v1, v2, v3);
  
			b1_val = b1_val/gamma_inv;
	        b2_val = b2_val/gamma_inv;
	        b3_val = b3_val/gamma_inv;
			const2 = (flcuda)2.0/((flcuda)1.0 + b1_val*b1_val + b2_val*b2_val + b3_val*b3_val);
			vv1 = v1[i];
			vv2 = v2[i];
			vv3 = v3[i];
			v1[i] = vv1 + const2*((vv2 - vv1*b3_val + vv3*b1_val)*b3_val - (vv3 + vv1*b2_val - vv2*b1_val)*b2_val);
			v2[i] = vv2 + const2*(-(vv1 + vv2*b3_val - vv3*b2_val)*b3_val + (vv3 + vv1*b2_val - vv2*b1_val)*b1_val); 
			v3[i] = vv3 + const2*((vv1 + vv2*b3_val - vv3*b2_val)*b2_val - (vv2 - vv1*b3_val + vv3*b1_val)*b1_val);

			//4. Half acceleration in the electric field
			//u(n+1/2) = u(n) + q*dt/2/m*E(n)
			v1[i] = v1[i] + e1_val;
			v2[i] = v2[i] + e2_val;
			v3[i] = v3[i] + e3_val;

			//5. Division by relativistic factor
			gamma = get_gamma_inv(i, v1, v2, v3);
		    v1[i] = v1[i]/gamma;
			v2[i] = v2[i]/gamma;
			v3[i] = v3[i]/gamma;
		}

}


__device__ flcuda get_field_e(flcuda x1, flcuda x3, flcuda* e_input, int component, Particles_struct cuda_specie)
{	
	int i_r=0;  // number of particle i cell 
	int k_z=0;  // number of particle k cell
	int counter;
	
	flcuda pi = 3.1415926535897932;
	flcuda dr = cuda_specie.dr;
	flcuda dz = cuda_specie.dz;
	flcuda r1, r2, r3; // temp variables for calculation
	flcuda dz1, dz2;   //  temp var.: width of k and k+1 cell 
	flcuda er =0;
	flcuda efi =0;
	flcuda ez =0;
	flcuda vol_1 =0; //  volume of i cell; Q/V, V - volume of elementary cell 
	flcuda vol_2 =0; //  volume of i+1 cell;
	
////////////////////////
	r1 = x1-0.5*dr;
	r3 = x1+0.5*dr;
///////////////////////

		switch (component)
	{
      case 1:
      {

    // weighting of E_r//
///////////////////////////////////////////////////
	//finding number of cell. example dr=0.5,  x1 = 0.7, i_r =0;!!
	 i_r = (int)ceil((x1-0.5*dr)/dr)-1;
	 k_z = (int)ceil((x3)/dz)-1;
	
	 vol_1 = pi*dz*dr*dr*(2*i_r+1);
	 vol_2 = pi*dz*dr*dr*(2*i_r+3);
	 dz1 = (k_z+1)*dz-x3;
	 dz2 = x3 - k_z*dz;
	 r2 = (i_r+1)*dr;
    ///////////////////////////////////////
  
    //weighting Er[i][k]//
   counter = get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 1);
   er = e_input[counter];
   er = er*(pi*dz1*(r2*r2-r1*r1))/vol_1;

	//weighting Er[i+1][k]//
   er = er + e_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 1)]*(pi*dz1*(r3*r3-r2*r2))/vol_2;

   //weighting Er[i][k+1]//
   er= er + e_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 1)]*(pi*dz2*(r2*r2-r1*r1))/vol_1;

   //weighting Er[i+1][k+1]//
   er = er + e_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 1)]*(pi*dz2*(r3*r3-r2*r2))/vol_2;
   
///////////////////////////////////////////////////////
   return er;
   }
   case 3:
   {
 
	
	     // weighting of E_z//
///////////////////////////////////////////////////////
//finding number of cell. example dz=0.5,  x3 = 0.7, z_k =0;!!
	i_r = (int)ceil((x1)/dr)-1;
	k_z = (int)ceil((x3-0.5*dz)/dz)-1;

///////////////////////////////////

   if(x1>dr)
	{
		vol_1 = pi*dz*dr*dr*2.0*(flcuda)i_r;
    }
   else
   {
	   vol_1 = pi*dz*dr*dr/4.0; //volume of first cell
   }
		  r2 = (i_r+0.5)*dr;
		  vol_2 = pi*dz*dr*dr*(2*i_r+2);
		  dz1 = (k_z+1.5)*dz - x3;
		  dz2 = x3 - (k_z+0.5)*dz;
		  //////////////////////////////////////

		   //weighting Ez[i][k]//
		   ez = ez + e_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 3)]*(pi*dz1*(r2*r2-r1*r1))/vol_1;

		  //weighting Ez[i+1][k]//
		   ez = ez + e_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 3)]*pi*dz1*(r3*r3-r2*r2)/vol_2;   

          //weighting Ez[i][k+1]//
		   ez = ez + e_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 3)]*pi*dz2*(r2*r2-r1*r1)/vol_1;
   
         //weighting Ez[i+1][k+1]//
		   ez = ez + e_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 3)]*pi*dz2*(r3*r3-r2*r2)/vol_2;    

	    return ez;
   }
   case 2:
	{

///////////////////////////////////////////////////////

	 // weighting of E_fi//
///////////////////////////////////////////////////////
 //finding number of cell. example dz=0.5,  x3 = 0.7, z_k =1;
	 i_r = (int)ceil((x1)/dr)-1;
     k_z = (int)ceil((x3)/dz)-1;
	
  if(x1>dr)
	{
		vol_1 = pi*dz*dr*dr*2.0*(flcuda)i_r;
    }
  else
  {
	 vol_1 = pi*dz*dr*dr/4.0; //volume of first cell
  }

		  r2 = (i_r+0.5)*dr;
		  vol_2 = pi*dz*dr*dr*(2*i_r+2);
		  dz1 = (k_z+1)*dz-x3;
		  dz2 = x3-k_z*dz;
		  //////////////////////////////////////
		  //weighting Efi[i][k]//
		  efi = efi + e_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 2)]*pi*dz1*(r2*r2 - r1*r1)/vol_1;

		  //weighting Efi[i+1][k]//
		   efi = efi + e_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 2)]*pi*dz1*(r3*r3-r2*r2)/vol_2;

          //weighting Efi[i][k+1]//
		   efi = efi + e_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 2)]*pi*dz2*(r2*r2-r1*r1)/vol_1;
   
         //weighting Efi[i+1][k+1]//
		   efi =efi + e_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 2)]*pi*dz2*(r3*r3-r2*r2)/vol_2;
		  return efi;
	}
	}
  		 
	return 0.0;
}

__device__ flcuda get_field_h(flcuda x1, flcuda x3, flcuda* h_input, int component, Particles_struct cuda_specie)
{
  int i_r = 0, k_z = 0;  
  flcuda r1, r2, r3, dz1, dz2, hr = 0.0, hfi = 0.0, hz = 0.0, 
	     vol_1 = 0.0, vol_2 = 0.0; //  volumes of i and i+1 cell; Q/V, V - volume of elementary cell 
	
  flcuda pi = 3.1415926535897932;
  flcuda dr = cuda_specie.dr;
  flcuda dz = cuda_specie.dz;

  ////////////////////////
  r1 = x1-0.5*dr;
  r3 = x1+0.5*dr;
  ///////////////////////

  switch (component)
  {
    case 3:
    {
      // weighting of H_z//
      i_r = (int)ceil((x1-0.5*dr)/dr)-1;
      k_z = (int)ceil((x3)/dz)-1;
      vol_1 = pi*dz*dr*dr*(2*i_r+1);
      vol_2 = pi*dz*dr*dr*(2*i_r+3);
      dz1 = (k_z+1)*dz-x3;
      dz2 = x3 - k_z*dz;
      r2 = (i_r+1)*dr;
      ///////////////////////////////////////  
      //weighting Hz[i][k]//
      hz = hz + h_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 6)]*(pi*dz1*(r2*r2-r1*r1))/vol_1;

      //weighting Hz[i+1][k]//
      hz = hz + h_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 6)]*(pi*dz1*(r3*r3-r2*r2))/vol_2;

      //weighting Hz[i][k+1]//
      hz = hz + h_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 6)]*(pi*dz2*(r3*r3-r2*r2))/vol_1;

      //weighting Hz[i+1][k+1]//
      hz = hz + h_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 6)]*(pi*dz2*(r3*r3-r2*r2))/vol_2;
      return hz;
    }
	case 1:
    {
      // weighting of Hr//
      i_r = (int)ceil((x1)/dr)-1;
      k_z = (int)ceil((x3-0.5*dz)/dz)-1;
      if(x1>dr)
      {
        vol_1 = pi*dz*dr*dr*2*i_r;
      }
      else 
      {
        vol_1 = pi*dz*dr*dr/4.0; //volume of first cell
      }
      r2 = (i_r+0.5)*dr;
      vol_2 = pi*dz*dr*dr*(2*i_r+2);
      dz1 = (k_z+1.5)*dz - x3;
      dz2 = x3 - (k_z+0.5)*dz;
      //////////////////////////////////////

      //weighting Hr[i][k]//
      hr = hr + h_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 4)]*(pi*dz1*(r2*r2-r1*r1))/vol_1;

      //weighting Hr[i+1][k]//
      hr = hr + h_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 4)]*pi*dz1*(r3*r3-r2*r2)/vol_2;   

      //weighting Hr[i][k+1]//
      hr = hr + h_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 4)]*pi*dz2*(r2*r2-r1*r1)/vol_1;
   
      //weighting Hr[i+1][k+1]//
      hr = hr + h_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 4)]*pi*dz2*(r3*r3-r2*r2)/vol_2; 
      return hr;
	}
	case 2:
    {
      // weighting of H_fi//
      i_r = (int)ceil((x1-0.5*dr)/dr)-1;
      k_z = (int)ceil((x3-0.5*dz)/dz)-1;
      r2 = (i_r+1)*dr;
      vol_1 = pi*dz*dr*dr*(2*i_r+1);
      vol_2 = pi*dz*dr*dr*(2*i_r+3);
      dz1 = (k_z+1.5)*dz-x3;
      dz2 = x3-(k_z+0.5)*dz;
      //weighting Hfi[i][k]//
      hfi = hfi + h_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 5)]*pi*dz1*(r2*r2-r1*r1)/vol_1;
	  //hfi = h_input[get_linear_coord(i_r, k_z, cuda_specie.grid_num3, 5)];
	  //hfi = h_input[1282];
	  //hfi = i_r;

      //weighting Hfi[i+1][k]//
      hfi = hfi + h_input[get_linear_coord(i_r+1, k_z, cuda_specie.grid_num3, 5)]*pi*dz1*(r3*r3-r2*r2)/vol_2;
		   
      //weighting Hfi[i][k+1]//
      hfi = hfi + h_input[get_linear_coord(i_r, k_z+1, cuda_specie.grid_num3, 5)]*dz2*pi*(r2*r2-r1*r1)/vol_1;
   
      //weighting Hfi[i+1][k+1]//
      hfi = hfi + h_input[get_linear_coord(i_r+1, k_z+1, cuda_specie.grid_num3, 5)]*pi*dz2*(r3*r3-r2*r2)/vol_2;  	
      return hfi;
    }
  }
  return 0.0;
}

__device__ int get_linear_coord(int index_r, int index_z, int ngrid_z, int component)
{
	//index components:
	// 1 - er, 2 - e_phi, 3 - e_z
	// 4 - hr, 5 - h_phi, 6 - h_z

	switch (component)
	{
		case 1:
		case 2:
		case 6:
			return (index_r * ngrid_z + index_z);
		case 3:
		case 4:
		case 5:
			return (index_r * (ngrid_z - 1) + index_z);
	}
					 
  return 0;
}

__device__ flcuda get_gamma(int i, flcuda* v1, flcuda* v2, flcuda* v3)
{
	return pow((flcuda)1.0 - (v1[i]*v1[i] + v2[i]*v2[i] + v3[i]*v3[i])/(flcuda)300000000.0/(flcuda)300000000.0,(flcuda)-0.5);
}

__device__ flcuda get_gamma_inv(int i, flcuda* v1, flcuda* v2, flcuda* v3)
{
	return pow((v1[i]*v1[i] + v2[i]*v2[i] + v3[i]*v3[i])/(flcuda)300000000.0/(flcuda)300000000.0 + (flcuda)1.0, (flcuda)0.5);	
}

